#include <stdio.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>
#include "nvidia.h"


int nvidia_init()
{
    int i;
    int deviceCount;
    struct hipDeviceProp_t prop;
    hipError_t err;

    hipGetDeviceCount(&deviceCount);

    fprintf(stderr, "------------------------------------------------------------------------------\n");
    fprintf(stderr, "  NVIDIA GPU Driver -- Found %i device(s).\n", deviceCount);

    for (i=0; i < deviceCount; i++)
    {
        fprintf(stderr, "\n\tDevice %i\n", i);

        err = hipGetDeviceProperties(&prop, i);
        if (err != hipSuccess)
        {
            fprintf(stderr, "\tError %i\n", err);
            continue;
        }

        fprintf(stderr, 
               "\tName: '%s'\n"
               "\tTotal Global Memory: %u\n"
               "\tShared Memory Per Block: %u\n"
               "\tRegisters Per Block: %i\n"
               "\tWarp Size: %i\n"
               "\tMemory Pitch: %u\n"
               "\tMaximum Threads Per Block: %i\n"
               "\tMaximum Size of Each Block Dimension: %5i %5i %5i\n"
               "\tMaximum Size of Each Grid Dimension:  %5i %5i %5i\n"
               "\tTotal Constant Memory: %i\n"
               "\tRevision: %i.%i\n"
               "\tClockrate: %iHz\n"
               "\tSize of Property Structure: %u\n", 
               prop.name,
               prop.totalGlobalMem,
               prop.sharedMemPerBlock,
               prop.regsPerBlock,
               prop.warpSize,
               prop.memPitch,
               prop.maxThreadsPerBlock,
               prop.maxThreadsDim[0],
               prop.maxThreadsDim[1],
               prop.maxThreadsDim[2],
               prop.maxGridSize[0],
               prop.maxGridSize[1],
               prop.maxGridSize[2],
               prop.totalConstMem,
               prop.major,
               prop.minor,
               prop.clockRate,
               sizeof(prop));

        hipSetDevice(i);
    }

    fprintf(stderr, "------------------------------------------------------------------------------\n");

    return 0;
}

